
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 100000000

__global__ void vector_add_kernel(float* out, float* a, float* b, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        out[i] = a[i] + b[i];
    }
}

void vector_add(float* C_h, float* A_h, float* B_h, int n) {
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
    hipMalloc((void**) &C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    vector_add_kernel<<<ceil(n/1024.0), 1024>>>(C_d, A_d, B_d, n);
    
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    float *a, *b, *out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    vector_add(out, a, b, N);
    printf("a[0] = %f\n", a[0]);
    printf("b[0] = %f\n", b[0]);
    printf("out[0] = %f\n", out[0]);
    free(a);
    free(b);
    free(out);
}